#include "hip/hip_runtime.h"
#include"core_class.h"
#include<hip/hip_cooperative_groups.h>

/*Test functions*/
using namespace cooperative_groups;

void check(hipError_t x) {
    fprintf(stderr, "%s\n", hipGetErrorString(x));
}
void copy_table_to_ram(simplex_table_cuda *st_d);

/*Simplex solver functions and kernel*/
int shared_memory_size=0;
__device__ void pivot_row_modifier(simplex_table_cuda *st_arr,float *pe_arr,int *p_row_arr,int *p_col_arr,char *completion_code,int blockIdx_x,int threadIdx_x)//ok check
{
    switch(threadIdx_x)
    {
        case 0:
        st_arr[blockIdx_x].r_id[p_row_arr[blockIdx_x]].basic=st_arr[blockIdx_x].c_id[p_col_arr[blockIdx_x]].basic;
        break;
        case 1:
        st_arr[blockIdx_x].r_id[p_row_arr[blockIdx_x]].id=st_arr[blockIdx_x].c_id[p_col_arr[blockIdx_x]].id;
        break;
        case 2:
        st_arr[blockIdx_x].r_id[p_row_arr[blockIdx_x]].rhs=st_arr[blockIdx_x].c_id[p_col_arr[blockIdx_x]].rhs;;
        break;
        case 3:
        st_arr[blockIdx_x].r_id[p_row_arr[blockIdx_x]].slack=st_arr[blockIdx_x].c_id[p_col_arr[blockIdx_x]].slack;
        break;
        case 4:
        st_arr[blockIdx_x].r_id[p_row_arr[blockIdx_x]].theta=st_arr[blockIdx_x].c_id[p_col_arr[blockIdx_x]].theta;
        break;
        default:
    }

    if(threadIdx_x<st_arr[blockIdx_x].basic_var_size_col)
    {   st_arr[blockIdx_x].basic_var[p_row_arr[blockIdx_x]*st_arr[blockIdx_x].basic_var_size_col+threadIdx_x]/=pe_arr[blockIdx_x];}
    else if(threadIdx_x>=st_arr[blockIdx_x].basic_var_size_col && threadIdx_x<(st_arr[blockIdx_x].basic_var_size_col+st_arr[blockIdx_x].slack_var_size_col))
    {
        int slack_col_index=threadIdx_x-st_arr[blockIdx_x].basic_var_size_col;
        st_arr[blockIdx_x].slack_var[p_row_arr[blockIdx_x]*st_arr[blockIdx_x].slack_var_size_col+slack_col_index]/=pe_arr[blockIdx_x];
    }
    else if(threadIdx_x==(st_arr[blockIdx_x].basic_var_size_col+st_arr[blockIdx_x].slack_var_size_col))
    {   st_arr[blockIdx_x].rhs[p_row_arr[blockIdx_x]]/=pe_arr[blockIdx_x];}
}

__global__ void rest_of_row_modifier(simplex_table_cuda *st_arr,float *pe_arr,int *p_row_arr,int *p_col_arr,char *completion_code,int loop_size,int no_of_tables)//ok check
{
    //row is blockIdx.y
    for(int a=0;a<loop_size;a++)
    {
        int index2=blockIdx.x*loop_size+a;
        if(index2<no_of_tables)
        {
            if(completion_code[index2]=='0')
            {
                pivot_row_modifier(st_arr,pe_arr,p_row_arr,p_col_arr,completion_code,index2,threadIdx.x);
            }
        }
    }
    __syncthreads();
    for(int a=0;a<loop_size;a++)
    {
        int index2=blockIdx.x*loop_size+a;
        if(index2<no_of_tables)
        {
            if(completion_code[index2]=='0')
            {
                if(threadIdx.x<st_arr[index2].basic_var_size_row)
                {
                    if(threadIdx.x!=p_row_arr[index2])//all row accept pivot row
                    {
                        float multiplying_element;
                        if(p_col_arr[index2]<st_arr[index2].basic_var_size_col)
                        {   multiplying_element=st_arr[index2].basic_var[threadIdx.x*st_arr[index2].basic_var_size_col+p_col_arr[index2]];}
                        else
                        {   
                            int index=p_col_arr[index2]-st_arr[index2].basic_var_size_col;
                            multiplying_element=st_arr[index2].slack_var[threadIdx.x*st_arr[index2].slack_var_size_col+index];
                        }
                        int basic_var_size_till_now_col=p_row_arr[index2]*st_arr[index2].basic_var_size_col;
                        int slack_var_size_till_now_col=p_row_arr[index2]*st_arr[index2].slack_var_size_col;
                        short col_size=st_arr[index2].basic_var_size_col+st_arr[index2].slack_var_size_col;
                        short slack_var_size_col=st_arr[index2].slack_var_size_col;
                        for(int blockIdx_y=0;blockIdx_y<col_size;blockIdx_y++)
                        {
                            if(blockIdx_y!=p_col_arr[index2])//dont touch the p col
                            {
                                if(blockIdx_y<st_arr[index2].basic_var_size_col)//basic_point
                                {   
                                    st_arr[index2].basic_var[threadIdx.x*st_arr[index2].basic_var_size_col+blockIdx_y]-=(multiplying_element*st_arr[index2].basic_var[basic_var_size_till_now_col+blockIdx_y]);
                                }
                                else if(blockIdx_y>=st_arr[index2].basic_var_size_col && blockIdx_y<(col_size))//slack_point
                                {
                                    int slack_col_index=blockIdx_y-st_arr[index2].basic_var_size_col;
                                    st_arr[index2].slack_var[threadIdx.x*slack_var_size_col+slack_col_index]-=(multiplying_element*st_arr[index2].slack_var[slack_var_size_till_now_col+slack_col_index]);
                                }
                            }
                        }
                        st_arr[index2].rhs[threadIdx.x]-=multiplying_element*st_arr[index2].rhs[p_row_arr[index2]];   
                    }
                }
            }
        }
    }
    __syncthreads();
    for(int a=0;a<loop_size;a++)
    {
        int index2=blockIdx.x*loop_size+a;
        if(index2<no_of_tables)
        {
            if(completion_code[index2]=='0')
            {
                if(threadIdx.x<st_arr[index2].basic_var_size_row && threadIdx.x!=p_row_arr[index2])
                {
                    if(p_col_arr[index2]<st_arr[index2].basic_var_size_col)
                    {
                        st_arr[index2].basic_var[threadIdx.x*st_arr[index2].basic_var_size_col+p_col_arr[index2]]=0;
                    }
                    else
                    {
                        int index=p_col_arr[index2]-st_arr[index2].basic_var_size_col;
                        st_arr[index2].slack_var[threadIdx.x*st_arr[index2].slack_var_size_col+index]=0;
                    }
                }
            }
        }
    }
}

void simplex_table_modifier(simplex_table_cuda *st_d_arr,int* row_with_negative_slack_d,float *pe_d_arr,int *p_row_arr_d,int *p_col_arr_d,char *completion_code_d,int largest_col,int largest_row,hipStream_t *stream1)//ok check
{
    //copy_table_to_ram(st_d_arr);//for testing
    //display_st(st_vec[11]);//for testing
    //int *row_with_negative_slack_test=(int*)malloc(sizeof(int)*st_vec.size());
    //int *p_row_test=(int*)malloc(sizeof(int)*st_vec.size());//for testing
    //int *p_col_test=(int*)malloc(sizeof(int)*st_vec.size());//for testing
    //float *pe_test=(float*)malloc(sizeof(float)*st_vec.size());//for testing
    //hipMemcpy(pe_test,pe_d_arr,sizeof(float)*st_vec.size(),hipMemcpyDeviceToHost);
    //hipMemcpy(p_col_test,p_col_arr_d,sizeof(int)*st_vec.size(),hipMemcpyDeviceToHost);//for testing
    //hipMemcpy(p_row_test,p_row_arr_d,sizeof(int)*st_vec.size(),hipMemcpyDeviceToHost);//for testing
    //hipMemcpy(row_with_negative_slack_test,row_with_negative_slack_d,sizeof(int)*st_vec.size(),hipMemcpyDeviceToHost);//for testing
    //cout<<"\nrow_with_negative_slack:"<<row_with_negative_slack_test[11];//for testing
    //cout<<"\np_row:"<<p_row_test[11]<<" p_col:"<<p_col_test[11]<<" pe:"<<pe_test[11]<<" ";//for testing
    //int gh;cin>>gh;

    largest_col++;//extr one for rhs
    //pivot row modifier
    if(largest_col<5)
    {   largest_col=5;}
    //rest of the row modifiew
    int no_of_threads_required;
    if(largest_col>largest_row)
    {   no_of_threads_required=largest_col;}
    else
    {   no_of_threads_required=largest_row;}
    int loop_size=10;
    int no_of_blocks=st_vec.size()/loop_size;
    if(no_of_blocks==0)
    {   no_of_blocks++;loop_size=st_vec.size();}
    else
    {
        if(st_vec.size()%loop_size)
        {   no_of_blocks++;}
    }
    rest_of_row_modifier<<<no_of_blocks,no_of_threads_required,shared_memory_size,*stream1>>>(st_d_arr,pe_d_arr,p_row_arr_d,p_col_arr_d,completion_code_d,loop_size,st_vec.size());
    //hipStreamSynchronize(*stream1);

    //copy_table_to_ram(st_d_arr);//for testing
    //display_st(st_vec[0]);//for testing
    //char *completion_code_test=(char*)malloc(sizeof(char)*st_vec.size());
    //hipMemcpy(completion_code_test,completion_code_d,sizeof(char)*st_vec.size(),hipMemcpyDeviceToHost);
    //if(completion_code_test[0]=='1')
    //{   cout<<"\n\nst solved!!!!!!";}
}

__global__ void termination_condition_checker_kernel(simplex_table_cuda *st_arr,char *completion_code,int no_of_tables)//ok check
{
    int index=blockIdx.x*32+threadIdx.x;
    if(index<no_of_tables)
    {
        if(completion_code[index]=='0')
        {
            bool status=true;
            for(int a=0;a<st_arr[index].basic_var_size_row;a++)
            {
                if(st_arr[index].r_id[a].slack)
                {   
                    if(st_arr[index].slack_var[a*st_arr[index].slack_var_size_col+st_arr[index].r_id[a].id-st_arr[index].basic_var_size_col]<0)
                    {   
                        if(st_arr[index].rhs[a]>=0)
                        {   status=false;break;}
                    }
                }
            }
            if(status)
            {   completion_code[index]='1';}
        }
    }
}

bool termination_condition_checker(simplex_table_cuda *st_d_arr,int largest_row,char *completion_code,char *completion_code_d,hipStream_t *stream1,int no_of_blocks)//ok check
{
    int termination_count=0;
    termination_condition_checker_kernel<<<no_of_blocks,32,shared_memory_size,*stream1>>>(st_d_arr,completion_code_d,st_vec.size());//block,thread
    hipStreamSynchronize(*stream1);
    hipMemcpy(completion_code,completion_code_d,sizeof(char)*st_vec.size(),hipMemcpyDeviceToHost);
    for(int a=0;a<st_vec.size();a++)
    {
        if(completion_code[a]!='0')
        {   termination_count++;}
    }
    //cout<<"\ntc: "<<termination_count<<" total: "<<st_vec.size();
    if(termination_count==st_vec.size())
    {   return true;}
    else
    {   return false;}
}

__global__ void find_row_with_negative_slack_and_p_col_kernel(simplex_table_cuda *st_arr,int *row_with_negative_slack,int *p_col_arr,char *completion_code,int no_of_table)//ok check
{
    int index=blockIdx.x*32+threadIdx.x;
    if(index<no_of_table)
    {
        if(completion_code[index]=='0')
        {
            row_with_negative_slack[index]=-1;
            for(int a=0;a<st_arr[index].basic_var_size_row;a++)
            {
                if(st_arr[index].r_id[a].slack)
                {
                    if(st_arr[index].slack_var[a*st_arr[index].slack_var_size_col+(st_arr[index].r_id[a].id-st_arr[index].basic_var_size_col)]<0 && st_arr[index].rhs[a]>=0)//originally it was just rhs>0, but now i feel it shouls be >=. Need further testing
                    {   row_with_negative_slack[index]=a;break;}
                }
            }
            if(row_with_negative_slack[index]==-1)
            {   completion_code[index]='5';}
            else
            {
                int col=-1;
                for(int a=0;a<st_arr[index].basic_var_size_col;a++)
                {
                    if(st_arr[index].basic_var[row_with_negative_slack[index]*st_arr[index].basic_var_size_col+a]>0)
                    {   col=a;break;}
                }
                if(col==-1)
                {
                    for(int a=0;a<st_arr[index].slack_var_size_col;a++)
                    {   
                        if(st_arr[index].slack_var[row_with_negative_slack[index]*st_arr[index].slack_var_size_col+a]>0)
                        {   col=a+st_arr[index].basic_var_size_col;break;}
                    }
                }
                if(col==-1)
                {   completion_code[index]='2';}
                p_col_arr[index]=col;
            }
        }
    }
}

void conflicting_data_finder(int st_table_index,vector<conflict_id> &conflict_id_vec)//ok check
{
    for(int a=0;a<st_vec[st_table_index]->r_id_size;a++)
    {
        if(st_vec[st_table_index]->r_id[a].slack && st_vec[st_table_index]->slack_var[a*st_vec[st_table_index]->slack_var_size_col+st_vec[st_table_index]->r_id[a].id-st_vec[st_table_index]->basic_var_size_col]<0 && st_vec[st_table_index]->rhs[a]>0)
        {   conflict_id_vec[st_table_index].id_vec.push_back(st_vec[st_table_index]->r_id[a].id-st_vec[st_table_index]->basic_var_size_col);}
    }
}

__device__ void check_for_cyclic_bug(int *p_col_arr_d,int *p_row_arr_d,buffer *buffer_obj_arr_d,char *completion_code_d,int index)
{
    if(completion_code_d[index]=='0')
    {
        if(buffer_obj_arr_d[index].small_index<buffer_obj_arr_d[index].small_size)
        {
            buffer_obj_arr_d[index].p_col_index_small[buffer_obj_arr_d[index].small_index]=p_col_arr_d[index];
            buffer_obj_arr_d[index].p_row_index_small[buffer_obj_arr_d[index].small_index]=p_row_arr_d[index];
            buffer_obj_arr_d[index].small_index++;
        }
        else
        {
            for(int a=0;a<buffer_obj_arr_d[index].small_size;a++)
            {
                if(a!=buffer_obj_arr_d[index].small_size-1)
                {   
                    buffer_obj_arr_d[index].p_col_index_small[a]=buffer_obj_arr_d[index].p_col_index_small[a+1];
                    buffer_obj_arr_d[index].p_row_index_small[a]=buffer_obj_arr_d[index].p_row_index_small[a+1];
                }
                else
                {   
                    buffer_obj_arr_d[index].p_col_index_small[a]=p_col_arr_d[index];
                    buffer_obj_arr_d[index].p_row_index_small[a]=p_row_arr_d[index];
                }
            }
        }
        if(buffer_obj_arr_d[index].large_index<buffer_obj_arr_d[index].large_size)
        {
            buffer_obj_arr_d[index].p_col_index[buffer_obj_arr_d[index].large_index]=p_col_arr_d[index];
            buffer_obj_arr_d[index].p_row_index[buffer_obj_arr_d[index].large_index]=p_row_arr_d[index];
            buffer_obj_arr_d[index].large_index++;
        }
        else
        {
            for(int b=0;b<buffer_obj_arr_d[index].large_index-buffer_obj_arr_d[index].small_size+1;b++)
            {
                int match=0;
                for(int c=0;c<buffer_obj_arr_d[index].small_size;c++)
                {
                    if(buffer_obj_arr_d[index].p_row_index[b+c]==buffer_obj_arr_d[index].p_row_index_small[c] && 
                        buffer_obj_arr_d[index].p_col_index[b+c]==buffer_obj_arr_d[index].p_col_index_small[c])
                    {   match++;}
                }
                if(match==buffer_obj_arr_d[index].small_size)
                {   completion_code_d[index]='4';break;}
            }
            if(completion_code_d[index]!='4')
            {
                for(int a=0;a<buffer_obj_arr_d[index].large_size;a++)
                {
                    if(a!=buffer_obj_arr_d[index].large_size-1)
                    {   
                        buffer_obj_arr_d[index].p_col_index[a]=buffer_obj_arr_d[index].p_col_index[a+1];
                        buffer_obj_arr_d[index].p_row_index[a]=buffer_obj_arr_d[index].p_row_index[a+1];
                    }
                    else
                    {   
                        buffer_obj_arr_d[index].p_col_index[a]=p_col_arr_d[index];
                        buffer_obj_arr_d[index].p_row_index[a]=p_row_arr_d[index];
                    }
                }
            }
        }
    }
}

__global__ void get_pivot_row_element_kernel(simplex_table_cuda *st_arr,int *p_row_arr,int *p_col_arr,float *pe_arr,char* completion_code,buffer *buffer_obj_arr_d,int no_of_table)//ok check
{
    int index=blockIdx.x*32+threadIdx.x;
    if(index<no_of_table)
    {
        if(completion_code[index]=='0')
        {
            p_row_arr[index]=-1;
            double smallest_value=-1;
            for(int a=0;a<st_arr[index].basic_var_size_row;a++)
            {
                if(st_arr[index].theta[a]>0)
                {
                    if(smallest_value==-1 || smallest_value>st_arr[index].theta[a])
                    {
                        smallest_value=st_arr[index].theta[a];
                        p_row_arr[index]=a;
                    }
                }
            }
            if(p_row_arr[index]<0)
            {   completion_code[index]='3';}
            else//get pe
            {
                if(p_col_arr[index]<st_arr[index].basic_var_size_col)
                {
                    pe_arr[index]=st_arr[index].basic_var[p_row_arr[index]*st_arr[index].basic_var_size_col+p_col_arr[index]];
                }
                else
                {
                    int slack_p_col=p_col_arr[index]-st_arr[index].basic_var_size_col;
                    pe_arr[index]=st_arr[index].slack_var[p_row_arr[index]*st_arr[index].slack_var_size_col+slack_p_col];
                }
            }
        }
        check_for_cyclic_bug(p_col_arr,p_row_arr,buffer_obj_arr_d,completion_code,index);
    }
}

__global__ void calc_theta_kernel(simplex_table_cuda *st_arr,int *row_with_negative_slack,int *pivote_col_index,char *completion_code)//ok check
{
    //if(threadIdx.x==0)
    //{   find_row_with_negative_slack_and_p_col_kernel(st_arr,row_with_negative_slack,pivote_col_index,completion_code,blockIdx.x);}
    //__syncthreads();
    if(completion_code[blockIdx.x]=='0' && threadIdx.x<st_arr[blockIdx.x].basic_var_size_row)
    {
        if(pivote_col_index[blockIdx.x]<st_arr[blockIdx.x].basic_var_size_col)
        {
            if(st_arr[blockIdx.x].basic_var[threadIdx.x*st_arr[blockIdx.x].basic_var_size_col+pivote_col_index[blockIdx.x]]==0)
            {   st_arr[blockIdx.x].theta[threadIdx.x]=0;}
            else
            {   st_arr[blockIdx.x].theta[threadIdx.x]=st_arr[blockIdx.x].rhs[threadIdx.x]/(double)st_arr[blockIdx.x].basic_var[threadIdx.x*st_arr[blockIdx.x].basic_var_size_col+pivote_col_index[blockIdx.x]];}
        }
        else
        {
            int temp_col_index=pivote_col_index[blockIdx.x]-st_arr[blockIdx.x].basic_var_size_col;
            if(st_arr[blockIdx.x].slack_var[threadIdx.x*st_arr[blockIdx.x].slack_var_size_col+temp_col_index]==0)
            {   st_arr[blockIdx.x].theta[threadIdx.x]=0;}
            else
            {   st_arr[blockIdx.x].theta[threadIdx.x]=st_arr[blockIdx.x].rhs[threadIdx.x]/(double)st_arr[blockIdx.x].slack_var[threadIdx.x*st_arr[blockIdx.x].slack_var_size_col+temp_col_index];}
        }
    }
    //__syncthreads();
    //int index=blockIdx.x*threads_per_block+threadIdx.x;
    //printf("\nindex: %d",index);
    //if(index<no_of_tables)
    //{   //printf("\nindex2: %d",index);
    //    get_pivot_row_element_kernel(st_arr,p_row_arr_d,pivote_col_index,pe_d_arr,completion_code,buffer_obj_arr_d,index);}
    //if(threadIdx.x==0)
    //{   get_pivot_row_element_kernel(st_arr,p_row_arr_d,pivote_col_index,pe_d_arr,completion_code,buffer_obj_arr_d,blockIdx.x);}
}

void free_everything_from_vram(simplex_table_cuda *st_d_arr,buffer* buffer_obj_arr_d)//ok check
{
    simplex_table_cuda *ram_arr=(simplex_table_cuda*)malloc(sizeof(simplex_table_cuda)*st_vec.size());
    hipMemcpy(ram_arr,st_d_arr,sizeof(simplex_table_cuda)*st_vec.size(),hipMemcpyDeviceToHost);

    buffer *buffer_obj_arr=(buffer*)malloc(sizeof(buffer)*st_vec.size());
    hipMemcpy(buffer_obj_arr,buffer_obj_arr_d,sizeof(buffer)*st_vec.size(),hipMemcpyDeviceToHost);
    for(int a=0;a<st_vec.size();a++)
    {
        hipFree(ram_arr[a].basic_var);
        hipFree(ram_arr[a].c_id);
        hipFree(ram_arr[a].r_id);
        hipFree(ram_arr[a].rhs);
        hipFree(ram_arr[a].slack_var);
        hipFree(ram_arr[a].theta);

        hipFree(buffer_obj_arr[a].p_col_index);        
        hipFree(buffer_obj_arr[a].p_row_index);
        hipFree(buffer_obj_arr[a].p_col_index_small);
        hipFree(buffer_obj_arr[a].p_row_index_small);
    }
    hipFree(st_d_arr);
    hipFree(buffer_obj_arr_d);
    free(ram_arr);
    free(buffer_obj_arr);
}

vector<conflict_id> pivot_element_finder(simplex_table_cuda *st_d_arr,buffer* buffer_obj_arr_d)
{
    int largest_row_size=0,largest_col_size=0;
    for(int a=0;a<st_vec.size();a++)
    {
        int col_size=st_vec[a]->basic_var_size_col+st_vec[a]->slack_var_size_col;
        if(largest_col_size<col_size)
        {   largest_col_size=col_size;}
        int row_size=st_vec[a]->basic_var_size_row;
        if(largest_row_size<row_size)
        {   largest_row_size=row_size;}
    }
    char *completion_code=(char*)malloc(sizeof(char)*st_vec.size());
    for(int a=0;a<st_vec.size();a++)
    {   completion_code[a]='0';}//0=not complete, 1=complete, 2=conflict_found, 3=bad_p_row, 4=cyclic_bug, 5=row_with_negative_element not found
    char *completion_code_d;
    hipMalloc(&completion_code_d,sizeof(char)*st_vec.size());
    hipMemcpy(completion_code_d,completion_code,sizeof(char)*st_vec.size(),hipMemcpyHostToDevice);

    vector<conflict_id> conflict_id_vec(st_vec.size());
    int *row_with_negative_slack_d;
    int *p_col_arr_d,*p_row_arr_d;
    hipMalloc(&p_col_arr_d,sizeof(int)*st_vec.size());
    hipMalloc(&p_row_arr_d,sizeof(int)*st_vec.size());
    
    hipStream_t stream1;
    hipStreamCreate(&stream1);
    float *pe_d_arr;
    //int iteration=0;
    hipMalloc(&pe_d_arr,sizeof(float)*st_vec.size());
    hipMalloc(&row_with_negative_slack_d,sizeof(float)*st_vec.size());
    int no_of_blocks=st_vec.size()/32,no_of_threads_rem=st_vec.size()%32;
    if(no_of_threads_rem!=0)
    {   no_of_blocks++;}
    do
    {
        //cout<<"\niteration: "<<iteration;
        find_row_with_negative_slack_and_p_col_kernel<<<no_of_blocks,32,shared_memory_size,stream1>>>(st_d_arr,row_with_negative_slack_d,p_col_arr_d,completion_code_d,st_vec.size());
        calc_theta_kernel<<<st_vec.size(),largest_row_size,shared_memory_size,stream1>>>(st_d_arr,row_with_negative_slack_d,p_col_arr_d,completion_code_d);
        get_pivot_row_element_kernel<<<no_of_blocks,32,shared_memory_size,stream1>>>(st_d_arr,p_row_arr_d,p_col_arr_d,pe_d_arr,completion_code_d,buffer_obj_arr_d,st_vec.size());
        //hipStreamSynchronize(stream1);
        simplex_table_modifier(st_d_arr,row_with_negative_slack_d,pe_d_arr,p_row_arr_d,p_col_arr_d,completion_code_d,largest_col_size,largest_row_size,&stream1);
        //iteration++;
    } 
    while(!termination_condition_checker(st_d_arr,largest_row_size,completion_code,completion_code_d,&stream1,no_of_blocks));
    hipStreamDestroy(stream1);
    hipMemcpy(completion_code,completion_code_d,sizeof(char)*st_vec.size(),hipMemcpyDeviceToHost);
    int complete=0,conflict=0;
    copy_table_to_ram(st_d_arr);
    for(int a=0;a<st_vec.size();a++)
    {
        conflict_id_vec[a].completion_code=completion_code[a];
        if(completion_code[a]=='1')
        {   complete++;}
        else if(completion_code[a]=='0')//engine shutdown before completing work.
        {   cout<<"\nERROR! engine shutdown before completing work. a: "<<a<<" completion_code: "<<completion_code[a];}
        else
        {   conflicting_data_finder(a,conflict_id_vec);conflict++;
            if(conflict_id_vec.at(a).id_vec.size()==0)
            {   
                cout<<"\nFailed to read conflict ids! "<<conflict_id_vec.at(a).id_vec.size();
                cout<<"\ncc: "<<completion_code[a];
                //display_st(st_vec[a]);
                //int *row_with_negative_slack=(int*)malloc(sizeof(int)*st_vec.size());
                //hipMemcpy(row_with_negative_slack,row_with_negative_slack_d,sizeof(int)*st_vec.size(),hipMemcpyDeviceToHost);
                //int *p_col_arr=(int*)malloc(sizeof(int)*st_vec.size());
                //hipMemcpy(p_col_arr,p_col_arr_d,sizeof(int)*st_vec.size(),hipMemcpyDeviceToHost);
                //int *p_row_arr=(int*)malloc(sizeof(int)*st_vec.size());
                //hipMemcpy(p_row_arr,p_row_arr_d,sizeof(int)*st_vec.size(),hipMemcpyDeviceToHost);
                //cout<<"\np_col: "<<p_col_arr[a]<<" p_row: "<<p_row_arr[a]<<" row_with-ve_slack: "<<row_with_negative_slack[a];
                //int gh;cin>>gh;
            }
        }
    }
    //cout<<"\ncomplete: "<<complete<<" conflict: "<<conflict;
    free(completion_code);
    hipFree(completion_code_d);
    hipFree(p_col_arr_d);
    hipFree(p_row_arr_d);
    hipFree(pe_d_arr);
    hipFree(row_with_negative_slack_d);
    
    return conflict_id_vec;
}

simplex_table_cuda* copy_table_to_vram(simplex_table_cuda *st_d_arr)//ok check
{
    for(int a=0;a<st_vec.size();a++)
    {
        st_d_arr[a].basic_var_size_col=st_vec[a]->basic_var_size_col;
        st_d_arr[a].basic_var_size_row=st_vec[a]->basic_var_size_row;
        hipMalloc(&st_d_arr[a].basic_var,sizeof(float)*st_d_arr[a].basic_var_size_col*st_d_arr[a].basic_var_size_row);
        hipMemcpy(st_d_arr[a].basic_var,st_vec[a]->basic_var,sizeof(float)*st_d_arr[a].basic_var_size_col*st_d_arr[a].basic_var_size_row,hipMemcpyHostToDevice);
        //free(st_vec[a]->basic_var);

        st_d_arr[a].c_id_size=st_vec[a]->c_id_size;
        hipMalloc(&st_d_arr[a].c_id,sizeof(id)*st_d_arr[a].c_id_size);
        hipMemcpy(st_d_arr[a].c_id,st_vec[a]->c_id,sizeof(id)*st_d_arr[a].c_id_size,hipMemcpyHostToDevice);
        //free(st_vec[a]->c_id);

        st_d_arr[a].r_id_size=st_vec[a]->r_id_size;
        hipMalloc(&st_d_arr[a].r_id,sizeof(id)*st_d_arr[a].r_id_size);
        hipMemcpy(st_d_arr[a].r_id,st_vec[a]->r_id,sizeof(id)*st_d_arr[a].r_id_size,hipMemcpyHostToDevice);
        //free(st_vec[a]->r_id);

        st_d_arr[a].slack_var_size_col=st_vec[a]->slack_var_size_col;
        st_d_arr[a].slack_var_size_row=st_vec[a]->slack_var_size_row;
        hipMalloc(&st_d_arr[a].slack_var,sizeof(float)*st_d_arr[a].slack_var_size_col*st_d_arr[a].slack_var_size_row);
        hipMemcpy(st_d_arr[a].slack_var,st_vec[a]->slack_var,sizeof(float)*st_d_arr[a].slack_var_size_col*st_d_arr[a].slack_var_size_row,hipMemcpyHostToDevice);
        //free(st_vec[a]->slack_var);

        st_d_arr[a].rhs_size=st_vec[a]->rhs_size;
        hipMalloc(&st_d_arr[a].rhs,sizeof(double)*st_d_arr[a].rhs_size);
        hipMemcpy(st_d_arr[a].rhs,st_vec[a]->rhs,sizeof(double)*st_d_arr[a].rhs_size,hipMemcpyHostToDevice);
        //free(st_vec[a]->rhs);
        
        hipMalloc(&st_d_arr[a].theta,sizeof(double)*st_d_arr[a].basic_var_size_row);
    }
    simplex_table_cuda *device_arr;
    hipMalloc(&device_arr,sizeof(simplex_table_cuda)*st_vec.size());
    hipMemcpy(device_arr,st_d_arr,sizeof(simplex_table_cuda)*st_vec.size(),hipMemcpyHostToDevice);
    free(st_d_arr);
    return device_arr;
}

buffer* copy_cyclic_bug_buffer_to_vram(buffer* buffer_obj_arr_d)
{
    for(int a=0;a<st_vec.size();a++)
    {
        buffer_obj_arr_d[a].large_size=4;
        buffer_obj_arr_d[a].small_size=2;
        buffer_obj_arr_d[a].small_index=0;
        buffer_obj_arr_d[a].large_index=0;

        hipMalloc(&buffer_obj_arr_d[a].p_row_index,sizeof(int)*buffer_obj_arr_d[a].large_size);
        hipMalloc(&buffer_obj_arr_d[a].p_col_index,sizeof(int)*buffer_obj_arr_d[a].large_size);
        hipMalloc(&buffer_obj_arr_d[a].p_row_index_small,sizeof(int)*buffer_obj_arr_d[a].small_size);
        hipMalloc(&buffer_obj_arr_d[a].p_col_index_small,sizeof(int)*buffer_obj_arr_d[a].small_size);
    }
    buffer *device_buffer;
    hipMalloc(&device_buffer,sizeof(buffer)*st_vec.size());
    hipMemcpy(device_buffer,buffer_obj_arr_d,sizeof(buffer)*st_vec.size(),hipMemcpyHostToDevice);
    free(buffer_obj_arr_d);
    return device_buffer;
}

void copy_table_to_ram(simplex_table_cuda *st_d_arr)//ok check
{
    simplex_table_cuda *ram_arr=(simplex_table_cuda*)malloc(sizeof(simplex_table_cuda)*st_vec.size());
    hipMemcpy(ram_arr,st_d_arr,sizeof(simplex_table_cuda)*st_vec.size(),hipMemcpyDeviceToHost);
    for(int a=0;a<st_vec.size();a++)
    {
        hipMemcpy(st_vec[a]->basic_var,ram_arr[a].basic_var,sizeof(float)*ram_arr[a].basic_var_size_col*ram_arr[a].basic_var_size_row,hipMemcpyDeviceToHost);
        hipMemcpy(st_vec[a]->c_id,ram_arr[a].c_id,sizeof(id)*ram_arr[a].c_id_size,hipMemcpyDeviceToHost);
        hipMemcpy(st_vec[a]->r_id,ram_arr[a].r_id,sizeof(id)*ram_arr[a].r_id_size,hipMemcpyDeviceToHost);
        hipMemcpy(st_vec[a]->slack_var,ram_arr[a].slack_var,sizeof(float)*ram_arr[a].slack_var_size_col*ram_arr[a].slack_var_size_row,hipMemcpyDeviceToHost);
        hipMemcpy(st_vec[a]->rhs,ram_arr[a].rhs,sizeof(double)*ram_arr[a].rhs_size,hipMemcpyDeviceToHost);
    }
    free(ram_arr);
}

vector<conflict_id> simplex_solver()
{
    //auto start = high_resolution_clock::now();
    //allocate the cyclic bug detecting buffer in vram
    buffer* buffer_obj_arr_d;
    buffer_obj_arr_d=(buffer*)malloc(sizeof(buffer)*st_vec.size());
    buffer_obj_arr_d=copy_cyclic_bug_buffer_to_vram(buffer_obj_arr_d);
    simplex_table_cuda *st_d_arr;
    st_d_arr=(simplex_table_cuda*)malloc(sizeof(simplex_table_cuda)*st_vec.size());
    st_d_arr=copy_table_to_vram(st_d_arr);
    vector<conflict_id> conflict_id_vec=pivot_element_finder(st_d_arr,buffer_obj_arr_d);
    free_everything_from_vram(st_d_arr,buffer_obj_arr_d);
    int complete=0,conflict=0;
    for(int a=0;a<conflict_id_vec.size();a++)
    {
        if(conflict_id_vec[a].id_vec.size()==0)
        {   complete++;}
        else
        {   conflict++;}
    }
    //cout<<"\ncomplete2: "<<complete<<" conflict2: "<<conflict;
    //auto end = high_resolution_clock::now();
    //auto duration = duration_cast<microseconds>(end - start); 
    //cout<<"\n\nduration= "<<duration.count()/pow(10,6)<<" sec";
    //int gh;cin>>gh;
    return conflict_id_vec;
}