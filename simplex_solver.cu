#include "hip/hip_runtime.h"
#include"core_class.h"
#include<thrust/device_vector.h>
#include<thrust/host_vector.h>

/*Test functions*/

void check(hipError_t x) {
    fprintf(stderr, "%s\n", hipGetErrorString(x));
}
void copy_table_to_ram(simplex_table_cuda *st_d);

/*Simplex solver functions and kernel*/
int shared_memory_size=0;
__global__ void pivot_row_modifier(simplex_table_cuda *st_arr,float *pe_arr,int *p_row_arr,int *p_col_arr,char *completion_code)//ok check
{
    if(completion_code[blockIdx.x]=='0')
    {
        switch(threadIdx.x)
        {
            case 0:
            st_arr[blockIdx.x].r_id[p_row_arr[blockIdx.x]].basic=st_arr[blockIdx.x].c_id[p_col_arr[blockIdx.x]].basic;
            break;
            case 1:
            st_arr[blockIdx.x].r_id[p_row_arr[blockIdx.x]].id=st_arr[blockIdx.x].c_id[p_col_arr[blockIdx.x]].id;
            break;
            case 2:
            st_arr[blockIdx.x].r_id[p_row_arr[blockIdx.x]].rhs=st_arr[blockIdx.x].c_id[p_col_arr[blockIdx.x]].rhs;;
            break;
            case 3:
            st_arr[blockIdx.x].r_id[p_row_arr[blockIdx.x]].slack=st_arr[blockIdx.x].c_id[p_col_arr[blockIdx.x]].slack;
            break;
            case 4:
            st_arr[blockIdx.x].r_id[p_row_arr[blockIdx.x]].theta=st_arr[blockIdx.x].c_id[p_col_arr[blockIdx.x]].theta;
            break;
            default:
        }

        if(threadIdx.x<st_arr[blockIdx.x].basic_var_size_col)
        {   st_arr[blockIdx.x].basic_var[p_row_arr[blockIdx.x]*st_arr[blockIdx.x].basic_var_size_col+threadIdx.x]/=pe_arr[blockIdx.x];}
        else if(threadIdx.x>=st_arr[blockIdx.x].basic_var_size_col && threadIdx.x<(st_arr[blockIdx.x].basic_var_size_col+st_arr[blockIdx.x].slack_var_size_col))
        {
            int slack_col_index=threadIdx.x-st_arr[blockIdx.x].basic_var_size_col;
            st_arr[blockIdx.x].slack_var[p_row_arr[blockIdx.x]*st_arr[blockIdx.x].slack_var_size_col+slack_col_index]/=pe_arr[blockIdx.x];
        }
        else if(threadIdx.x==(st_arr[blockIdx.x].basic_var_size_col+st_arr[blockIdx.x].slack_var_size_col))
        {   st_arr[blockIdx.x].rhs[p_row_arr[blockIdx.x]]/=pe_arr[blockIdx.x];}
    }
}

__global__ void rest_of_row_modifier(simplex_table_cuda *st_arr,int *p_row_arr,int *p_col_arr,/*float *multiplying_element_matrix,*/int largest_row,char *completion_code)//ok check
{
    //row is blockIdx.y
    if(completion_code[blockIdx.x]=='0')
    {
        if(threadIdx.x<st_arr[blockIdx.x].basic_var_size_row)
        {
            if(threadIdx.x!=p_row_arr[blockIdx.x])//all row accept pivot row
            {
                if(blockIdx.y!=p_col_arr[blockIdx.x])
                {
                    float multiplying_element;
                    if(p_col_arr[blockIdx.x]<st_arr[blockIdx.x].basic_var_size_col)
                    {   multiplying_element=st_arr[blockIdx.x].basic_var[threadIdx.x*st_arr[blockIdx.x].basic_var_size_col+p_col_arr[blockIdx.x]];}
                    else
                    {   
                        int index=p_col_arr[blockIdx.x]-st_arr[blockIdx.x].basic_var_size_col;
                        multiplying_element=st_arr[blockIdx.x].slack_var[threadIdx.x*st_arr[blockIdx.x].slack_var_size_col+index];
                    }
                    if(blockIdx.y<(st_arr[blockIdx.x].basic_var_size_col+st_arr[blockIdx.x].slack_var_size_col))
                    {
                        if(blockIdx.y<st_arr[blockIdx.x].basic_var_size_col)//basic_point
                        {   
                            st_arr[blockIdx.x].basic_var[threadIdx.x*st_arr[blockIdx.x].basic_var_size_col+blockIdx.y]-=(multiplying_element*st_arr[blockIdx.x].basic_var[p_row_arr[blockIdx.x]*st_arr[blockIdx.x].basic_var_size_col+blockIdx.y]);
                        }
                        else if(blockIdx.y>=st_arr[blockIdx.x].basic_var_size_col && blockIdx.y<(st_arr[blockIdx.x].basic_var_size_col+st_arr[blockIdx.x].slack_var_size_col))//slack_point
                        {
                            int slack_col_index=blockIdx.y-st_arr[blockIdx.x].basic_var_size_col;
                            st_arr[blockIdx.x].slack_var[threadIdx.x*st_arr[blockIdx.x].slack_var_size_col+slack_col_index]-=(multiplying_element*st_arr[blockIdx.x].slack_var[p_row_arr[blockIdx.x]*st_arr[blockIdx.x].slack_var_size_col+slack_col_index]);
                        }
                    }
                    else if(blockIdx.y==st_arr[blockIdx.x].basic_var_size_col+st_arr[blockIdx.x].slack_var_size_col)//rhs
                    {
                        st_arr[blockIdx.x].rhs[threadIdx.x]-=multiplying_element*st_arr[blockIdx.x].rhs[p_row_arr[blockIdx.x]];   
                    }
                }
            }
        }
    }
}

__global__ void p_col_modifier(simplex_table_cuda *st_arr,int *p_row_arr,int *p_col_arr,char *completion_code)
{
    if(completion_code[blockIdx.x]=='0')
    {
        if(threadIdx.x<st_arr[blockIdx.x].basic_var_size_row && threadIdx.x!=p_row_arr[blockIdx.x])
        {
            if(p_col_arr[blockIdx.x]<st_arr[blockIdx.x].basic_var_size_col)
            {
                st_arr[blockIdx.x].basic_var[threadIdx.x*st_arr[blockIdx.x].basic_var_size_col+p_col_arr[blockIdx.x]]=0;
            }
            else
            {
                int index=p_col_arr[blockIdx.x]-st_arr[blockIdx.x].basic_var_size_col;
                st_arr[blockIdx.x].slack_var[threadIdx.x*st_arr[blockIdx.x].slack_var_size_col+index]=0;
            }
        }
    }
}

void simplex_table_modifier(simplex_table_cuda *st_d_arr,int* row_with_negative_slack_d,float *pe_d_arr,int *p_row_arr_d,int *p_col_arr_d,char *completion_code_d,int largest_col,int largest_row,hipStream_t *stream1)//ok check
{
    //copy_table_to_ram(st_d_arr);//for testing
    //display_st(st_vec[11]);//for testing
    //int *row_with_negative_slack_test=(int*)malloc(sizeof(int)*st_vec.size());
    //int *p_row_test=(int*)malloc(sizeof(int)*st_vec.size());//for testing
    //int *p_col_test=(int*)malloc(sizeof(int)*st_vec.size());//for testing
    //float *pe_test=(float*)malloc(sizeof(float)*st_vec.size());//for testing
    //hipMemcpy(pe_test,pe_d_arr,sizeof(float)*st_vec.size(),hipMemcpyDeviceToHost);
    //hipMemcpy(p_col_test,p_col_arr_d,sizeof(int)*st_vec.size(),hipMemcpyDeviceToHost);//for testing
    //hipMemcpy(p_row_test,p_row_arr_d,sizeof(int)*st_vec.size(),hipMemcpyDeviceToHost);//for testing
    //hipMemcpy(row_with_negative_slack_test,row_with_negative_slack_d,sizeof(int)*st_vec.size(),hipMemcpyDeviceToHost);//for testing
    //cout<<"\nrow_with_negative_slack:"<<row_with_negative_slack_test[11];//for testing
    //cout<<"\np_row:"<<p_row_test[11]<<" p_col:"<<p_col_test[11]<<" pe:"<<pe_test[11]<<" ";//for testing
    //int gh;cin>>gh;

    largest_col++;//extr one for rhs
    //pivot row modifier
    if(largest_col<5)
    {   largest_col=5;}
    pivot_row_modifier<<<st_vec.size(),largest_col,shared_memory_size,*stream1>>>(st_d_arr,pe_d_arr,p_row_arr_d,p_col_arr_d,completion_code_d);
    //rest of the row modifiew

    dim3 block_vec(st_vec.size(),largest_col,1);
    rest_of_row_modifier<<<block_vec,largest_row,shared_memory_size,*stream1>>>(st_d_arr,p_row_arr_d,p_col_arr_d,/*multiplying_element_matrix,*/largest_row,completion_code_d);
    p_col_modifier<<<st_vec.size(),largest_row,shared_memory_size,*stream1>>>(st_d_arr,p_row_arr_d,p_col_arr_d,completion_code_d);
    hipStreamSynchronize(*stream1);

    //copy_table_to_ram(st_d_arr);//for testing
    //display_st(st_vec[0]);//for testing
    //char *completion_code_test=(char*)malloc(sizeof(char)*st_vec.size());
    //hipMemcpy(completion_code_test,completion_code_d,sizeof(char)*st_vec.size(),hipMemcpyDeviceToHost);
    //if(completion_code_test[0]=='1')
    //{   cout<<"\n\nst solved!!!!!!";}
}

__global__ void termination_condition_checker_kernel(simplex_table_cuda *st_arr,char *completion_code,int no_of_tables)//ok check
{
    int index=blockIdx.x*32+threadIdx.x;
    if(index<no_of_tables)
    {
        if(completion_code[index]=='0')
        {
            bool status=true;
            for(int a=0;a<st_arr[index].basic_var_size_row;a++)
            {
                if(st_arr[index].r_id[a].slack)
                {   
                    if(st_arr[index].slack_var[a*st_arr[index].slack_var_size_col+st_arr[index].r_id[a].id-st_arr[index].basic_var_size_col]<0)
                    {   
                        if(st_arr[index].rhs[a]>=0)
                        {   status=false;break;}
                    }
                }
            }
            if(status)
            {   completion_code[index]='1';}
        }
    }
}

bool termination_condition_checker(simplex_table_cuda *st_d_arr,int largest_row,char *completion_code,char *completion_code_d,hipStream_t *stream1,int no_of_blocks)//ok check
{
    int termination_count=0;
    termination_condition_checker_kernel<<<no_of_blocks,32,shared_memory_size,*stream1>>>(st_d_arr,completion_code_d,st_vec.size());//block,thread
    hipStreamSynchronize(*stream1);
    hipMemcpy(completion_code,completion_code_d,sizeof(char)*st_vec.size(),hipMemcpyDeviceToHost);
    for(int a=0;a<st_vec.size();a++)
    {
        if(completion_code[a]!='0')
        {   termination_count++;}
    }
    if(termination_count==st_vec.size())
    {   return true;}
    else
    {   return false;}
}

__global__ void find_row_with_negative_slack_and_p_col_kernel(simplex_table_cuda *st_arr,int *row_with_negative_slack,int *p_col_arr,char *completion_code,int no_of_tables)//ok check
{
    int index=blockIdx.x*32+threadIdx.x;
    if(index<no_of_tables)
    {
        if(completion_code[index]=='0')
        {
            row_with_negative_slack[index]=-1;
            for(int a=0;a<st_arr[index].basic_var_size_row;a++)
            {
                if(st_arr[index].r_id[a].slack)
                {
                    if(st_arr[index].slack_var[a*st_arr[index].slack_var_size_col+(st_arr[index].r_id[a].id-st_arr[index].basic_var_size_col)]<0 && st_arr[index].rhs[a]>=0)//originally it was just rhs>0, but now i feel it shouls be >=. Need further testing
                    {   row_with_negative_slack[index]=a;break;}
                }
            }
            if(row_with_negative_slack[index]==-1)
            {   completion_code[index]='5';}
            else
            {
                int col=-1;
                for(int a=0;a<st_arr[index].basic_var_size_col;a++)
                {
                    if(st_arr[index].basic_var[row_with_negative_slack[index]*st_arr[index].basic_var_size_col+a]>0)
                    {   col=a;break;}
                }
                if(col==-1)
                {
                    for(int a=0;a<st_arr[index].slack_var_size_col;a++)
                    {   
                        if(st_arr[index].slack_var[row_with_negative_slack[index]*st_arr[index].slack_var_size_col+a]>0)
                        {   col=a+st_arr[index].basic_var_size_col;break;}
                    }
                }
                if(col==-1)
                {   completion_code[index]='2';}
                p_col_arr[index]=col;
            }
        }
    }
}

void conflicting_data_finder(int st_table_index,vector<conflict_id> &conflict_id_vec)//ok check
{
    for(int a=0;a<st_vec[st_table_index]->r_id_size;a++)
    {
        if(st_vec[st_table_index]->r_id[a].slack && st_vec[st_table_index]->slack_var[a*st_vec[st_table_index]->slack_var_size_col+st_vec[st_table_index]->r_id[a].id-st_vec[st_table_index]->basic_var_size_col]<0 && st_vec[st_table_index]->rhs[a]>0)
        {   conflict_id_vec[st_table_index].id_vec.push_back(st_vec[st_table_index]->r_id[a].id-st_vec[st_table_index]->basic_var_size_col);}
    }
}

__global__ void calc_theta_kernel(simplex_table_cuda *st_arr,int *pivote_col_index,char *completion_code)//ok check
{
    if(completion_code[blockIdx.x]=='0' && threadIdx.x<st_arr[blockIdx.x].basic_var_size_row)
    {
        if(pivote_col_index[blockIdx.x]<st_arr[blockIdx.x].basic_var_size_col)
        {
            if(st_arr[blockIdx.x].basic_var[threadIdx.x*st_arr[blockIdx.x].basic_var_size_col+pivote_col_index[blockIdx.x]]==0)
            {   st_arr[blockIdx.x].theta[threadIdx.x]=0;}
            else
            {   st_arr[blockIdx.x].theta[threadIdx.x]=st_arr[blockIdx.x].rhs[threadIdx.x]/(double)st_arr[blockIdx.x].basic_var[threadIdx.x*st_arr[blockIdx.x].basic_var_size_col+pivote_col_index[blockIdx.x]];}
        }
        else
        {
            int temp_col_index=pivote_col_index[blockIdx.x]-st_arr[blockIdx.x].basic_var_size_col;
            if(st_arr[blockIdx.x].slack_var[threadIdx.x*st_arr[blockIdx.x].slack_var_size_col+temp_col_index]==0)
            {   st_arr[blockIdx.x].theta[threadIdx.x]=0;}
            else
            {   st_arr[blockIdx.x].theta[threadIdx.x]=st_arr[blockIdx.x].rhs[threadIdx.x]/(double)st_arr[blockIdx.x].slack_var[threadIdx.x*st_arr[blockIdx.x].slack_var_size_col+temp_col_index];}
        }
    }
}

__global__ void get_pivot_row_element_kernel(simplex_table_cuda *st_arr,int *p_row_arr,int *p_col_arr,float *pe_arr,char* completion_code,int no_of_tables)//ok check
{
    int index=blockIdx.x*32+threadIdx.x;
    if(index<no_of_tables)
    {
        if(completion_code[index]=='0')
        {
            p_row_arr[index]=-1;
            double smallest_value=-1;
            for(int a=0;a<st_arr[index].basic_var_size_row;a++)
            {
                if(st_arr[index].theta[a]>0)
                {
                    if(smallest_value==-1 || smallest_value>st_arr[index].theta[a])
                    {
                        smallest_value=st_arr[index].theta[a];
                        p_row_arr[index]=a;
                    }
                }
            }
            if(p_row_arr[index]<0)
            {   completion_code[index]='3';}
            else//get pe
            {
                if(p_col_arr[index]<st_arr[index].basic_var_size_col)
                {
                    pe_arr[index]=st_arr[index].basic_var[p_row_arr[index]*st_arr[index].basic_var_size_col+p_col_arr[index]];
                }
                else
                {
                    int slack_p_col=p_col_arr[index]-st_arr[index].basic_var_size_col;
                    pe_arr[index]=st_arr[index].slack_var[p_row_arr[index]*st_arr[index].slack_var_size_col+slack_p_col];
                }
            }
        }
    }
}

void check_for_cyclic_bug(int *p_col_arr_d,int *p_row_arr_d,vector<buffer> &buffer_obj_vec,simplex_table_cuda *st_d_arr,char *completion_code,char *completion_code_d)//need to be checked. The algorithm used here is new and much better as it should have exceptionally low false positives.
{
    int size_large=4,size_small=2;//large must be disible by small
    int *p_col_arr,*p_row_arr;
    p_col_arr=(int*)malloc(sizeof(int)*st_vec.size());
    hipMemcpy(p_col_arr,p_col_arr_d,sizeof(int)*st_vec.size(),hipMemcpyDeviceToHost);
    p_row_arr=(int*)malloc(sizeof(int)*st_vec.size());
    hipMemcpy(p_row_arr,p_row_arr_d,sizeof(int)*st_vec.size(),hipMemcpyDeviceToHost);
    hipMemcpy(completion_code,completion_code_d,sizeof(char)*st_vec.size(),hipMemcpyDeviceToHost);
    bool cc_changed=false;
    for(int a=0;a<st_vec.size();a++)
    {
        if(completion_code[a]=='0')
        {
            if(buffer_obj_vec[a].p_col_index_small.size()<size_small)
            {
                buffer_obj_vec[a].p_col_index_small.push_back(p_col_arr[a]);
                buffer_obj_vec[a].p_row_index_small.push_back(p_row_arr[a]);
            }
            else
            {
                buffer_obj_vec[a].p_col_index_small.push_back(p_col_arr[a]);
                buffer_obj_vec[a].p_row_index_small.push_back(p_row_arr[a]);
                buffer_obj_vec[a].p_col_index_small.erase(buffer_obj_vec[a].p_col_index_small.begin());
                buffer_obj_vec[a].p_row_index_small.erase(buffer_obj_vec[a].p_row_index_small.begin());   
            }
            if(buffer_obj_vec[a].p_col_index.size()<size_large)
            {
                buffer_obj_vec[a].p_col_index.push_back(p_col_arr[a]);
                buffer_obj_vec[a].p_row_index.push_back(p_row_arr[a]);
            }
            else
            {
                for(int b=0;b<buffer_obj_vec[a].p_row_index.size()-size_small+1;b++)
                {
                    int match=0;
                    for(int c=0;c<buffer_obj_vec[a].p_col_index_small.size();c++)
                    {
                        if(buffer_obj_vec[a].p_row_index[b+c]==buffer_obj_vec[a].p_row_index_small[c] && 
                           buffer_obj_vec[a].p_col_index[b+c]==buffer_obj_vec[a].p_col_index_small[c])
                        {   match++;}
                    }
                    if(match==size_small)
                    {   
                        completion_code[a]='4';
                        cc_changed=true;
                        break;
                    }
                }
                if(completion_code[a]!='4')
                {
                    buffer_obj_vec[a].p_col_index.push_back(p_col_arr[a]);
                    buffer_obj_vec[a].p_row_index.push_back(p_row_arr[a]);
                    buffer_obj_vec[a].p_col_index.erase(buffer_obj_vec[a].p_col_index.begin());
                    buffer_obj_vec[a].p_row_index.erase(buffer_obj_vec[a].p_row_index.begin());
                }
            }
        }
    }
    if(cc_changed)
    {   hipMemcpy(completion_code_d,completion_code,sizeof(char)*st_vec.size(),hipMemcpyHostToDevice);}
    free(p_row_arr);
    free(p_col_arr);
}

void free_simplex_table_from_vram(simplex_table_cuda *st_d_arr)//ok check
{
    simplex_table_cuda *ram_arr=(simplex_table_cuda*)malloc(sizeof(simplex_table_cuda)*st_vec.size());
    hipMemcpy(ram_arr,st_d_arr,sizeof(simplex_table_cuda)*st_vec.size(),hipMemcpyDeviceToHost);
    for(int a=0;a<st_vec.size();a++)
    {
        hipFree(ram_arr[a].basic_var);
        hipFree(ram_arr[a].c_id);
        hipFree(ram_arr[a].r_id);
        hipFree(ram_arr[a].rhs);
        hipFree(ram_arr[a].slack_var);
        hipFree(ram_arr[a].theta);
    }
    hipFree(st_d_arr);
    free(ram_arr);
}

vector<conflict_id> pivot_element_finder(simplex_table_cuda *st_d_arr)
{
    int largest_row_size=0,largest_col_size=0;
    for(int a=0;a<st_vec.size();a++)
    {
        int col_size=st_vec[a]->basic_var_size_col+st_vec[a]->slack_var_size_col;
        if(largest_col_size<col_size)
        {   largest_col_size=col_size;}
        int row_size=st_vec[a]->basic_var_size_row;
        if(largest_row_size<row_size)
        {   largest_row_size=row_size;}
    }
    char *completion_code=(char*)malloc(sizeof(char)*st_vec.size());
    for(int a=0;a<st_vec.size();a++)
    {   completion_code[a]='0';}//0=not complete, 1=complete, 2=conflict_found, 3=bad_p_row, 4=cyclic_bug, 5=row_with_negative_element not found
    char *completion_code_d;
    hipMalloc(&completion_code_d,sizeof(char)*st_vec.size());
    hipMemcpy(completion_code_d,completion_code,sizeof(char)*st_vec.size(),hipMemcpyHostToDevice);

    vector<conflict_id> conflict_id_vec(st_vec.size());
    int *row_with_negative_slack_d;
    int *p_col_arr_d,*p_row_arr_d;
    hipMalloc(&p_col_arr_d,sizeof(int)*st_vec.size());
    hipMalloc(&p_row_arr_d,sizeof(int)*st_vec.size());
    vector<buffer> buffer_obj_vec(st_vec.size());
    hipStream_t stream1;
    hipStreamCreate(&stream1);
    float *pe_d_arr;
    //int iteration=0;
    hipMalloc(&pe_d_arr,sizeof(float)*st_vec.size());
    hipMalloc(&row_with_negative_slack_d,sizeof(float)*st_vec.size());
    int no_of_blocks=st_vec.size()/32,no_of_threads_rem=st_vec.size()%32;
    if(no_of_threads_rem!=0)
    {   no_of_blocks++;}
    do
    {
        //cout<<"\niteration: "<<iteration;
        find_row_with_negative_slack_and_p_col_kernel<<<no_of_blocks,32,shared_memory_size,stream1>>>(st_d_arr,row_with_negative_slack_d,p_col_arr_d,completion_code_d,st_vec.size());
        calc_theta_kernel<<<st_vec.size(),largest_row_size,shared_memory_size,stream1>>>(st_d_arr,p_col_arr_d,completion_code_d);
        get_pivot_row_element_kernel<<<no_of_blocks,32,shared_memory_size,stream1>>>(st_d_arr,p_row_arr_d,p_col_arr_d,pe_d_arr,completion_code_d,st_vec.size());
        hipStreamSynchronize(stream1);
        check_for_cyclic_bug(p_col_arr_d,p_row_arr_d,buffer_obj_vec,st_d_arr,completion_code,completion_code_d);
        simplex_table_modifier(st_d_arr,row_with_negative_slack_d,pe_d_arr,p_row_arr_d,p_col_arr_d,completion_code_d,largest_col_size,largest_row_size,&stream1);
        //iteration++;
    } 
    while(!termination_condition_checker(st_d_arr,largest_row_size,completion_code,completion_code_d,&stream1,no_of_blocks));
    hipStreamDestroy(stream1);
    buffer_obj_vec.clear();
    hipMemcpy(completion_code,completion_code_d,sizeof(char)*st_vec.size(),hipMemcpyDeviceToHost);
    int complete=0,conflict=0;
    copy_table_to_ram(st_d_arr);
    for(int a=0;a<st_vec.size();a++)
    {
        conflict_id_vec[a].completion_code=completion_code[a];
        if(completion_code[a]=='1')
        {   complete++;}
        else if(completion_code[a]=='0')//engine shutdown before completing work.
        {   cout<<"\nERROR! engine shutdown before completing work. a: "<<a<<" completion_code: "<<completion_code[a];}
        else
        {   conflicting_data_finder(a,conflict_id_vec);conflict++;
            if(conflict_id_vec.at(a).id_vec.size()==0)
            {   
                cout<<"\nFailed to read conflict ids! "<<conflict_id_vec.at(a).id_vec.size();
                cout<<"\ncc: "<<completion_code[a];
                //display_st(st_vec[a]);
                //int *row_with_negative_slack=(int*)malloc(sizeof(int)*st_vec.size());
                //hipMemcpy(row_with_negative_slack,row_with_negative_slack_d,sizeof(int)*st_vec.size(),hipMemcpyDeviceToHost);
                //int *p_col_arr=(int*)malloc(sizeof(int)*st_vec.size());
                //hipMemcpy(p_col_arr,p_col_arr_d,sizeof(int)*st_vec.size(),hipMemcpyDeviceToHost);
                //int *p_row_arr=(int*)malloc(sizeof(int)*st_vec.size());
                //hipMemcpy(p_row_arr,p_row_arr_d,sizeof(int)*st_vec.size(),hipMemcpyDeviceToHost);
                //cout<<"\np_col: "<<p_col_arr[a]<<" p_row: "<<p_row_arr[a]<<" row_with-ve_slack: "<<row_with_negative_slack[a];
                //int gh;cin>>gh;
            }
        }
    }
    //cout<<"\ncomplete: "<<complete<<" conflict: "<<conflict;
    free(completion_code);
    hipFree(completion_code_d);
    hipFree(p_col_arr_d);
    hipFree(p_row_arr_d);
    hipFree(pe_d_arr);
    hipFree(row_with_negative_slack_d);
    
    return conflict_id_vec;
}

simplex_table_cuda* copy_table_to_vram(simplex_table_cuda *st_d_arr)//ok check
{
    for(int a=0;a<st_vec.size();a++)
    {
        st_d_arr[a].basic_var_size_col=st_vec[a]->basic_var_size_col;
        st_d_arr[a].basic_var_size_row=st_vec[a]->basic_var_size_row;
        hipMalloc(&st_d_arr[a].basic_var,sizeof(float)*st_d_arr[a].basic_var_size_col*st_d_arr[a].basic_var_size_row);
        hipMemcpy(st_d_arr[a].basic_var,st_vec[a]->basic_var,sizeof(float)*st_d_arr[a].basic_var_size_col*st_d_arr[a].basic_var_size_row,hipMemcpyHostToDevice);
        //free(st_vec[a]->basic_var);

        st_d_arr[a].c_id_size=st_vec[a]->c_id_size;
        hipMalloc(&st_d_arr[a].c_id,sizeof(id)*st_d_arr[a].c_id_size);
        hipMemcpy(st_d_arr[a].c_id,st_vec[a]->c_id,sizeof(id)*st_d_arr[a].c_id_size,hipMemcpyHostToDevice);
        //free(st_vec[a]->c_id);

        st_d_arr[a].r_id_size=st_vec[a]->r_id_size;
        hipMalloc(&st_d_arr[a].r_id,sizeof(id)*st_d_arr[a].r_id_size);
        hipMemcpy(st_d_arr[a].r_id,st_vec[a]->r_id,sizeof(id)*st_d_arr[a].r_id_size,hipMemcpyHostToDevice);
        //free(st_vec[a]->r_id);

        st_d_arr[a].slack_var_size_col=st_vec[a]->slack_var_size_col;
        st_d_arr[a].slack_var_size_row=st_vec[a]->slack_var_size_row;
        hipMalloc(&st_d_arr[a].slack_var,sizeof(float)*st_d_arr[a].slack_var_size_col*st_d_arr[a].slack_var_size_row);
        hipMemcpy(st_d_arr[a].slack_var,st_vec[a]->slack_var,sizeof(float)*st_d_arr[a].slack_var_size_col*st_d_arr[a].slack_var_size_row,hipMemcpyHostToDevice);
        //free(st_vec[a]->slack_var);

        st_d_arr[a].rhs_size=st_vec[a]->rhs_size;
        hipMalloc(&st_d_arr[a].rhs,sizeof(double)*st_d_arr[a].rhs_size);
        hipMemcpy(st_d_arr[a].rhs,st_vec[a]->rhs,sizeof(double)*st_d_arr[a].rhs_size,hipMemcpyHostToDevice);
        //free(st_vec[a]->rhs);
        
        hipMalloc(&st_d_arr[a].theta,sizeof(double)*st_d_arr[a].basic_var_size_row);
    }
    simplex_table_cuda *device_arr;
    hipMalloc(&device_arr,sizeof(simplex_table_cuda)*st_vec.size());
    hipMemcpy(device_arr,st_d_arr,sizeof(simplex_table_cuda)*st_vec.size(),hipMemcpyHostToDevice);
    free(st_d_arr);
    return device_arr;
}

void copy_table_to_ram(simplex_table_cuda *st_d_arr)//ok check
{
    simplex_table_cuda *ram_arr=(simplex_table_cuda*)malloc(sizeof(simplex_table_cuda)*st_vec.size());
    hipMemcpy(ram_arr,st_d_arr,sizeof(simplex_table_cuda)*st_vec.size(),hipMemcpyDeviceToHost);
    for(int a=0;a<st_vec.size();a++)
    {
        hipMemcpy(st_vec[a]->basic_var,ram_arr[a].basic_var,sizeof(float)*ram_arr[a].basic_var_size_col*ram_arr[a].basic_var_size_row,hipMemcpyDeviceToHost);
        hipMemcpy(st_vec[a]->c_id,ram_arr[a].c_id,sizeof(id)*ram_arr[a].c_id_size,hipMemcpyDeviceToHost);
        hipMemcpy(st_vec[a]->r_id,ram_arr[a].r_id,sizeof(id)*ram_arr[a].r_id_size,hipMemcpyDeviceToHost);
        hipMemcpy(st_vec[a]->slack_var,ram_arr[a].slack_var,sizeof(float)*ram_arr[a].slack_var_size_col*ram_arr[a].slack_var_size_row,hipMemcpyDeviceToHost);
        hipMemcpy(st_vec[a]->rhs,ram_arr[a].rhs,sizeof(double)*ram_arr[a].rhs_size,hipMemcpyDeviceToHost);
    }
    free(ram_arr);
}

vector<conflict_id> simplex_solver()
{
    //auto start = high_resolution_clock::now();
    simplex_table_cuda *st_d_arr;
    st_d_arr=(simplex_table_cuda*)malloc(sizeof(simplex_table_cuda)*st_vec.size());
    st_d_arr=copy_table_to_vram(st_d_arr);
    vector<conflict_id> conflict_id_vec=pivot_element_finder(st_d_arr);
    free_simplex_table_from_vram(st_d_arr);
    int complete=0,conflict=0;
    for(int a=0;a<conflict_id_vec.size();a++)
    {
        if(conflict_id_vec[a].id_vec.size()==0)
        {   complete++;}
        else
        {   conflict++;}
    }
    //cout<<"\ncomplete2: "<<complete<<" conflict2: "<<conflict;
    //auto end = high_resolution_clock::now();
    //auto duration = duration_cast<microseconds>(end - start); 
    //cout<<"\n\nduration= "<<duration.count()/pow(10,6)<<" sec";
    //int gh;cin>>gh;
    return conflict_id_vec;
}